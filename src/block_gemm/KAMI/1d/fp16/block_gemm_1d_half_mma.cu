#include <mma.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define WARP_SIZE 32

using namespace nvcuda;

#ifndef M_BLOCK
#define M_BLOCK 128
#define N_BLOCK 128
#define K_BLOCK 128
#endif

#ifndef NUM_RANK_BLOCK
#define NUM_RANK_BLOCK 4
#endif

#ifndef NUM_ALLOC_RANK_BLOCK
#define NUM_ALLOC_RANK_BLOCK NUM_RANK_BLOCK
#endif

#define WMMA_M_BLOCK 16
#define WMMA_N_BLOCK 8
#define WMMA_K_BLOCK 16

#define SUB_M_BLOCK (M_BLOCK / NUM_RANK_BLOCK)
#define SUB_N_BLOCK (N_BLOCK / NUM_RANK_BLOCK)
#define SUB_K_BLOCK (K_BLOCK / NUM_RANK_BLOCK)

#ifndef NUM_PIPE_BLOCK
#define NUM_PIPE_BLOCK 1
#endif

const int shmem_size_block_gemm = SUB_N_BLOCK * K_BLOCK / NUM_PIPE_BLOCK * sizeof(half);

#define NUM_ITER 1000

#define THREADS_PER_BLOCK (NUM_RANK_BLOCK * WARP_SIZE)

#define BLOCKS_PER_GRID 16384

__device__ __forceinline__ void mma_m16n8k16_fp16(uint32_t *acc, uint32_t *frag_a, uint32_t *frag_b)
{
    uint32_t const *A = reinterpret_cast<uint32_t const *>(&frag_a[0]);
    uint32_t const *B = reinterpret_cast<uint32_t const *>(&frag_b[0]);
    uint32_t *C = reinterpret_cast<uint32_t *>(&acc[0]);

    asm volatile(
        "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16"
        " { %0, %1 }, "
        " { %2, %3, %4, %5 }, "
        " { %6, %7 }, "
        " { %8, %9 };"
        : "=r"(C[0]), "=r"(C[1])
        : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]),
          "r"(B[0]), "r"(B[1]),
          "r"(C[0]), "r"(C[1]));
}

__global__ void block_gemm_1d_half_mma(const half *A, const half *B, half *C, const half alpha, const half beta)
{
    extern __shared__ half shmem_b[];

    const unsigned int warp_id = threadIdx.x >> 5;
    const unsigned int lane_id = threadIdx.x & 31;

    if (warp_id >= NUM_RANK_BLOCK)
    {
        return;
    }

    uint32_t (*b_compute)[K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK][2 * WARP_SIZE] = reinterpret_cast<uint32_t (*)[K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK][2 * WARP_SIZE]>(shmem_b);

    uint32_t a_frags[SUB_M_BLOCK / WMMA_M_BLOCK][K_BLOCK / WMMA_K_BLOCK][4];
    uint32_t b_frags[SUB_N_BLOCK / WMMA_N_BLOCK][K_BLOCK / WMMA_K_BLOCK][2];

    uint32_t c_frags[SUB_M_BLOCK / WMMA_M_BLOCK][N_BLOCK / WMMA_N_BLOCK][2];

    for (int i = 0; i < SUB_M_BLOCK / WMMA_M_BLOCK; ++i)
    {
        for (int j = 0; j < K_BLOCK / WMMA_K_BLOCK; ++j)
        {
            int group_id = lane_id >> 2;
            int thread_in_group = lane_id % 4;
            int row_upper = group_id;
            int row_lower = group_id + 8;
            int col_left = thread_in_group * 2;
            int col_right = thread_in_group * 2 + 8;

            int global_offset_upper_left = warp_id * (SUB_M_BLOCK * K_BLOCK) + (i * WMMA_M_BLOCK + row_upper) * K_BLOCK + j * WMMA_K_BLOCK + col_left;
            int global_offset_lower_left = warp_id * (SUB_M_BLOCK * K_BLOCK) + (i * WMMA_M_BLOCK + row_lower) * K_BLOCK + j * WMMA_K_BLOCK + col_left;

            int global_offset_upper_right = warp_id * (SUB_M_BLOCK * K_BLOCK) + (i * WMMA_M_BLOCK + row_upper) * K_BLOCK + j * WMMA_K_BLOCK + col_right;
            int global_offset_lower_right = warp_id * (SUB_M_BLOCK * K_BLOCK) + (i * WMMA_M_BLOCK + row_lower) * K_BLOCK + j * WMMA_K_BLOCK + col_right;

            a_frags[i][j][0] = *(uint32_t *)(&(A[global_offset_upper_left]));
            a_frags[i][j][1] = *(uint32_t *)(&(A[global_offset_lower_left]));
            a_frags[i][j][2] = *(uint32_t *)(&(A[global_offset_upper_right]));
            a_frags[i][j][3] = *(uint32_t *)(&(A[global_offset_lower_right]));
        }
    }

    for (int i = 0; i < SUB_N_BLOCK / WMMA_N_BLOCK; ++i)
    {
        for (int j = 0; j < K_BLOCK / WMMA_K_BLOCK; ++j)
        {
            int row = lane_id >> 2;
            int col_left = (lane_id % 4) * 2;
            int col_right = (lane_id % 4) * 2 + 8;
            int global_offset_left = warp_id * (SUB_N_BLOCK * K_BLOCK) + (i * WMMA_N_BLOCK + row) * K_BLOCK + j * WMMA_K_BLOCK + col_left;
            int global_offset_right = warp_id * (SUB_N_BLOCK * K_BLOCK) + (i * WMMA_N_BLOCK + row) * K_BLOCK + j * WMMA_K_BLOCK + col_right;

            b_frags[i][j][0] = *(uint32_t *)(&(B[global_offset_left]));
            b_frags[i][j][1] = *(uint32_t *)(&(B[global_offset_right]));
        }
    }

    for (int i = 0; i < SUB_M_BLOCK / WMMA_M_BLOCK; ++i)
    {
        for (int j = 0; j < N_BLOCK / WMMA_N_BLOCK; ++j)
        {
            c_frags[i][j][0] = 0.0;
            c_frags[i][j][1] = 0.0;
        }
    }

    __syncthreads();

    for (int idx_iter = 0; idx_iter < NUM_ITER; ++idx_iter)
    {
        for (int idx_stage = 0; idx_stage < NUM_RANK_BLOCK; ++idx_stage)
        {
            for (int idx_pipe = 0; idx_pipe < NUM_PIPE_BLOCK; ++idx_pipe)
            {
                if (warp_id == idx_stage)
                {
                    for (int i = 0; i < SUB_N_BLOCK / WMMA_N_BLOCK; i++)
                    {
                        for (int j = 0; j < K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK; j++)
                        {
                            int real_j = j + idx_pipe * (K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK);

                            b_compute[i][j][lane_id * 2 + 0] = b_frags[i][real_j][0];
                            b_compute[i][j][lane_id * 2 + 1] = b_frags[i][real_j][1];
                        }
                    }
                }

                __syncthreads();

#pragma unroll
                for (int i = 0; i < SUB_M_BLOCK / WMMA_M_BLOCK; ++i)
                {
#pragma unroll
                    for (int j = 0; j < SUB_N_BLOCK / WMMA_N_BLOCK; ++j)
                    {
#pragma unroll
                        for (int k = 0; k < K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK; ++k)
                        {
                            int real_k = k + idx_pipe * (K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK);
                            mma_m16n8k16_fp16(c_frags[i][j + idx_stage * (SUB_N_BLOCK / WMMA_N_BLOCK)], a_frags[i][real_k], &b_compute[j][k][lane_id * 2]);
                        }
                    }
                }

                __syncthreads();
            }
        }
    }

    for (int i = 0; i < SUB_M_BLOCK / WMMA_M_BLOCK; i++)
    {
        for (int j = 0; j < N_BLOCK / WMMA_N_BLOCK; j++)
        {
            int block_row = i;
            int block_col = j;

            int group_id = lane_id >> 2;
            int thread_in_group = lane_id % 4;
            int row_upper = group_id;
            int row_lower = group_id + 8;
            int col = thread_in_group * 2;
            int global_row_upper = block_row * WMMA_M_BLOCK + row_upper;
            int global_row_lower = block_row * WMMA_M_BLOCK + row_lower;
            int global_col = block_col * WMMA_N_BLOCK + col;

            int global_offset_upper = global_row_upper * N_BLOCK + global_col + warp_id * (SUB_M_BLOCK * N_BLOCK);
            int global_offset_lower = global_row_lower * N_BLOCK + global_col + warp_id * (SUB_M_BLOCK * N_BLOCK);

            *(uint32_t *)(&(C[global_offset_upper])) = c_frags[i][j][0];
            *(uint32_t *)(&(C[global_offset_lower])) = c_frags[i][j][1];
        }
    }
}

int main(int argc, char *argv[])
{
    int device_id = 0;
    hipDeviceProp_t prop;
    hipSetDevice(device_id);
    hipGetDeviceProperties(&prop, device_id);
    std::cout << "GPU " << prop.pciDeviceID << " Model: " << prop.name << std::endl;

    half *h_A = (half *)malloc(M_BLOCK * K_BLOCK * sizeof(half));
    half *h_B = (half *)malloc(K_BLOCK * N_BLOCK * sizeof(half));

    for (int i = 0; i < M_BLOCK * K_BLOCK; i++)
    {

        h_A[i] = 1;
    }

    for (int i = 0; i < K_BLOCK * N_BLOCK; i++)
    {
        h_B[i] = rand() % 3;
    }

    half *d_A, *d_B;
    half *d_C;

    hipMalloc(&d_A, M_BLOCK * K_BLOCK * sizeof(half));
    hipMalloc(&d_B, K_BLOCK * N_BLOCK * sizeof(half));
    hipMalloc(&d_C, M_BLOCK * N_BLOCK * sizeof(half));

    hipMemcpy(d_A, h_A, M_BLOCK * K_BLOCK * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K_BLOCK * N_BLOCK * sizeof(half), hipMemcpyHostToDevice);
    hipMemset(d_C, 0, M_BLOCK * N_BLOCK * sizeof(half));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    std::cout << "Launching kernel with " << BLOCKS_PER_GRID << " blocks and "
              << NUM_ALLOC_RANK_BLOCK << " threads per block and " << shmem_size_block_gemm << " bytes of shared memory" << std::endl;

    hipFuncSetAttribute(reinterpret_cast<const void*>(block_gemm_1d_half_mma), hipFuncAttributeMaxDynamicSharedMemorySize, shmem_size_block_gemm);
    hipEventRecord(start);
    block_gemm_1d_half_mma<<<BLOCKS_PER_GRID, NUM_ALLOC_RANK_BLOCK * WARP_SIZE, shmem_size_block_gemm>>>(d_A, d_B, d_C, 1.0, 0.0);
    hipDeviceSynchronize();
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    long long numOpsPerMatrix = 2LL * M_BLOCK * N_BLOCK * K_BLOCK;
    long long totalOps = numOpsPerMatrix * BLOCKS_PER_GRID * NUM_ITER;
    double gflops = static_cast<double>(totalOps) / (milliseconds * 1e6);
    double tflops = gflops / 1000.0f;

    std::cout << "Execution time: " << milliseconds << " ms" << std::endl;
    std::cout << "Performance: " << gflops << " GFLOPS (" << tflops << " TFLOPS)" << std::endl;
    std::cout << "[hemeng_log],1d," << M_BLOCK << "," << N_BLOCK << "," << K_BLOCK << "," << NUM_RANK_BLOCK << "," << NUM_ALLOC_RANK_BLOCK << "," << tflops << "," << NUM_ALLOC_RANK_BLOCK * 32 << std::endl;

    half *h_C = (half *)malloc(M_BLOCK * N_BLOCK * sizeof(half));
    hipMemcpy(h_C, d_C, M_BLOCK * N_BLOCK * sizeof(half), hipMemcpyDeviceToHost);

    double *h_C_ref = (double *)malloc(M_BLOCK * N_BLOCK * sizeof(double));

    for (int i = 0; i < M_BLOCK; i++)
    {
        for (int j = 0; j < M_BLOCK; j++)
        {
            h_C_ref[i * N_BLOCK + j] = 0;
            for (int k = 0; k < K_BLOCK; k++)
            {
                h_C_ref[i * N_BLOCK + j] += (double)h_A[i * K_BLOCK + k] * (double)h_B[j * K_BLOCK + k];
            }
        }
    }

    int error = 0;
    for (int i = 0; i < M_BLOCK * N_BLOCK; i++)
    {
        if (fabs((double)h_C[i] / NUM_ITER - h_C_ref[i]) > 1e-6)
        {

            error = 1;
            break;
        }
    }

    if (!error)
    {
        std::cout << "Validation successful!" << std::endl;
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
