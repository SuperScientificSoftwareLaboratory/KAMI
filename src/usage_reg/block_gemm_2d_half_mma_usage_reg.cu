
#include <mma.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define WARP_SIZE 32

using namespace nvcuda;

#ifndef M_BLOCK
#define M_BLOCK 64
#define N_BLOCK 64
#define K_BLOCK 64
#endif

#ifndef NUM_RANK_BLOCK
#define NUM_RANK_BLOCK 2
#endif

#define NUM_RANK_ALL_BLOCK (NUM_RANK_BLOCK * NUM_RANK_BLOCK)

#ifndef NUM_ALLOC_RANK_BLOCK
#define NUM_ALLOC_RANK_BLOCK NUM_RANK_ALL_BLOCK
#endif

#define WMMA_M_BLOCK 16
#define WMMA_N_BLOCK 8
#define WMMA_K_BLOCK 16

#define SUB_M_BLOCK (M_BLOCK / NUM_RANK_BLOCK)
#define SUB_N_BLOCK (N_BLOCK / NUM_RANK_BLOCK)
#define SUB_K_BLOCK (K_BLOCK / NUM_RANK_BLOCK)

#define NUM_PIPE_BLOCK 1

const int shmem_size_block_gemm = ((M_BLOCK * (SUB_K_BLOCK / NUM_PIPE_BLOCK)) + (K_BLOCK / NUM_PIPE_BLOCK * SUB_N_BLOCK)) * sizeof(half);

#ifndef NUM_ITER
#define NUM_ITER 1
#endif

#define THREADS_PER_BLOCK (NUM_RANK_ALL_BLOCK * WARP_SIZE)

#define BLOCKS_PER_GRID 1

__device__ __forceinline__ void mma_m16n8k16_fp16(uint32_t *acc, uint32_t *frag_a, uint32_t *frag_b)
{
    uint32_t const *A = reinterpret_cast<uint32_t const *>(&frag_a[0]);
    uint32_t const *B = reinterpret_cast<uint32_t const *>(&frag_b[0]);
    uint32_t *C = reinterpret_cast<uint32_t *>(&acc[0]);

    asm volatile(
        "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16"
        " { %0, %1 }, "
        " { %2, %3, %4, %5 }, "
        " { %6, %7 }, "
        " { %8, %9 };"
        : "=r"(C[0]), "=r"(C[1])
        : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]),
          "r"(B[0]), "r"(B[1]),
          "r"(C[0]), "r"(C[1]));
}

__global__ void block_gemm_2d_half_mma(const half *A, const half *B, half *C, const half alpha, const half beta)
{
    extern __shared__ __align__(16) half shmem[];

    half *shmem_a = shmem;
    half *shmem_b = shmem + M_BLOCK * SUB_K_BLOCK / NUM_PIPE_BLOCK;

    const unsigned int warp_id = threadIdx.x >> 5;

    const unsigned int lane_id = threadIdx.x & 31;

    const unsigned int warp_row = warp_id / NUM_RANK_BLOCK;

    const unsigned int warp_col = warp_id % NUM_RANK_BLOCK;

    if (warp_row >= NUM_RANK_BLOCK || warp_col >= NUM_RANK_BLOCK)
    {
        return;
    }

    uint32_t a_frags[SUB_M_BLOCK / WMMA_M_BLOCK][SUB_K_BLOCK / WMMA_K_BLOCK][4];

    uint32_t b_frags[SUB_N_BLOCK / WMMA_N_BLOCK][SUB_K_BLOCK / WMMA_K_BLOCK][2];

    uint32_t a_frags_compute[SUB_M_BLOCK / WMMA_M_BLOCK][SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK][4];

    uint32_t b_frags_compute[SUB_N_BLOCK / WMMA_N_BLOCK][SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK][2];

    uint32_t c_frags[SUB_M_BLOCK / WMMA_M_BLOCK][SUB_N_BLOCK / WMMA_N_BLOCK][2];

    for (int i = 0; i < SUB_M_BLOCK / WMMA_M_BLOCK; ++i)
    {
        for (int j = 0; j < SUB_K_BLOCK / WMMA_K_BLOCK; ++j)
        {
            int block_row = warp_row * (SUB_M_BLOCK / WMMA_M_BLOCK) + i;
            int block_col = warp_col * (SUB_K_BLOCK / WMMA_K_BLOCK) + j;
            int real_row_upper = block_row * WMMA_M_BLOCK + (lane_id >> 2);
            int real_row_lower = real_row_upper + 8;
            int real_col_left = block_col * WMMA_K_BLOCK + (lane_id % 4) * 2;
            int real_col_right = real_col_left + 8;
            int global_offset_upper_left = real_row_upper * K_BLOCK + real_col_left;
            int global_offset_lower_left = real_row_lower * K_BLOCK + real_col_left;
            int global_offset_upper_right = real_row_upper * K_BLOCK + real_col_right;
            int global_offset_lower_right = real_row_lower * K_BLOCK + real_col_right;

            a_frags[i][j][0] = *(uint32_t *)(A + global_offset_upper_left);
            a_frags[i][j][1] = *(uint32_t *)(A + global_offset_lower_left);
            a_frags[i][j][2] = *(uint32_t *)(A + global_offset_upper_right);
            a_frags[i][j][3] = *(uint32_t *)(A + global_offset_lower_right);
        }
    }

    for (int i = 0; i < SUB_N_BLOCK / WMMA_N_BLOCK; ++i)
    {
        for (int j = 0; j < SUB_K_BLOCK / WMMA_K_BLOCK; ++j)
        {
            int block_row = warp_col * (SUB_N_BLOCK / WMMA_N_BLOCK) + i;
            int block_col = warp_row * (SUB_K_BLOCK / WMMA_K_BLOCK) + j;
            int real_row = block_row * WMMA_N_BLOCK + (lane_id >> 2);
            int real_col_left = block_col * WMMA_K_BLOCK + (lane_id % 4) * 2;
            int real_col_right = real_col_left + 8;
            int global_offset_left = real_row * K_BLOCK + real_col_left;
            int global_offset_right = real_row * K_BLOCK + real_col_right;

            b_frags[i][j][0] = *(uint32_t *)(B + global_offset_left);
            b_frags[i][j][1] = *(uint32_t *)(B + global_offset_right);
        }
    }

    for (int i = 0; i < SUB_M_BLOCK / WMMA_M_BLOCK; ++i)
    {
        for (int j = 0; j < SUB_N_BLOCK / WMMA_N_BLOCK; ++j)
        {
            c_frags[i][j][0] = 0.0;
            c_frags[i][j][1] = 0.0;
        }
    }

    __syncthreads();

    for (int idx_iter = 0; idx_iter < NUM_ITER; ++idx_iter)
    {
        for (int idx_stage = 0; idx_stage < NUM_RANK_BLOCK; ++idx_stage)
        {
            for (int idx_pipe = 0; idx_pipe < NUM_PIPE_BLOCK; ++idx_pipe)
            {

                if (warp_col == idx_stage)
                {
                    for (int i = 0; i < SUB_M_BLOCK / WMMA_M_BLOCK; i++)
                    {
                        for (int j = 0; j < SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK; j++)
                        {
                            int real_j = j + idx_pipe * (SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK);
                            *(uint32_t *)(shmem_a + lane_id * 8 + 0 + (i * (SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK) + j) * WMMA_M_BLOCK * WMMA_K_BLOCK + warp_row * ((SUB_M_BLOCK) * (SUB_K_BLOCK / NUM_PIPE_BLOCK))) = a_frags[i][real_j][0];
                            *(uint32_t *)(shmem_a + lane_id * 8 + 2 + (i * (SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK) + j) * WMMA_M_BLOCK * WMMA_K_BLOCK + warp_row * ((SUB_M_BLOCK) * (SUB_K_BLOCK / NUM_PIPE_BLOCK))) = a_frags[i][real_j][1];
                            *(uint32_t *)(shmem_a + lane_id * 8 + 4 + (i * (SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK) + j) * WMMA_M_BLOCK * WMMA_K_BLOCK + warp_row * ((SUB_M_BLOCK) * (SUB_K_BLOCK / NUM_PIPE_BLOCK))) = a_frags[i][real_j][2];
                            *(uint32_t *)(shmem_a + lane_id * 8 + 6 + (i * (SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK) + j) * WMMA_M_BLOCK * WMMA_K_BLOCK + warp_row * ((SUB_M_BLOCK) * (SUB_K_BLOCK / NUM_PIPE_BLOCK))) = a_frags[i][real_j][3];
                        }
                    }
                }

                if (warp_row == idx_stage)
                {
                    for (int i = 0; i < SUB_N_BLOCK / WMMA_N_BLOCK; i++)
                    {
                        for (int j = 0; j < SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK; j++)
                        {
                            int real_j = j + idx_pipe * (SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK);
                            *(uint32_t *)(shmem_b + lane_id * 4 + 0 + (i * (SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK) + j) * WMMA_K_BLOCK * WMMA_N_BLOCK + warp_col * ((SUB_K_BLOCK / NUM_PIPE_BLOCK) * (SUB_N_BLOCK))) = b_frags[i][real_j][0];
                            *(uint32_t *)(shmem_b + lane_id * 4 + 2 + (i * (SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK) + j) * WMMA_K_BLOCK * WMMA_N_BLOCK + warp_col * ((SUB_K_BLOCK / NUM_PIPE_BLOCK) * (SUB_N_BLOCK))) = b_frags[i][real_j][1];
                        }
                    }
                }

                __syncthreads();

                for (int i = 0; i < SUB_M_BLOCK / WMMA_M_BLOCK; i++)
                {
                    for (int j = 0; j < SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK; j++)
                    {
                        a_frags_compute[i][j][0] = *(uint32_t *)(shmem_a + lane_id * 8 + 0 + (i * (SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK) + j) * WMMA_M_BLOCK * WMMA_K_BLOCK + warp_row * ((SUB_M_BLOCK) * (SUB_K_BLOCK / NUM_PIPE_BLOCK)));
                        a_frags_compute[i][j][1] = *(uint32_t *)(shmem_a + lane_id * 8 + 2 + (i * (SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK) + j) * WMMA_M_BLOCK * WMMA_K_BLOCK + warp_row * ((SUB_M_BLOCK) * (SUB_K_BLOCK / NUM_PIPE_BLOCK)));
                        a_frags_compute[i][j][2] = *(uint32_t *)(shmem_a + lane_id * 8 + 4 + (i * (SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK) + j) * WMMA_M_BLOCK * WMMA_K_BLOCK + warp_row * ((SUB_M_BLOCK) * (SUB_K_BLOCK / NUM_PIPE_BLOCK)));
                        a_frags_compute[i][j][3] = *(uint32_t *)(shmem_a + lane_id * 8 + 6 + (i * (SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK) + j) * WMMA_M_BLOCK * WMMA_K_BLOCK + warp_row * ((SUB_M_BLOCK) * (SUB_K_BLOCK / NUM_PIPE_BLOCK)));
                    }
                }

                for (int i = 0; i < SUB_N_BLOCK / WMMA_N_BLOCK; i++)
                {
                    for (int j = 0; j < SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK; j++)
                    {
                        b_frags_compute[i][j][0] = *(uint32_t *)(shmem_b + lane_id * 4 + 0 + (i * (SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK) + j) * WMMA_K_BLOCK * WMMA_N_BLOCK + warp_col * ((SUB_K_BLOCK / NUM_PIPE_BLOCK) * (SUB_N_BLOCK)));
                        b_frags_compute[i][j][1] = *(uint32_t *)(shmem_b + lane_id * 4 + 2 + (i * (SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK) + j) * WMMA_K_BLOCK * WMMA_N_BLOCK + warp_col * ((SUB_K_BLOCK / NUM_PIPE_BLOCK) * (SUB_N_BLOCK)));
                    }
                }

                __syncthreads();

                for (int i = 0; i < SUB_M_BLOCK / WMMA_M_BLOCK; ++i)
                {
                    for (int j = 0; j < SUB_N_BLOCK / WMMA_N_BLOCK; ++j)
                    {
                        for (int k = 0; k < SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_BLOCK; ++k)
                        {
                            mma_m16n8k16_fp16(c_frags[i][j], a_frags_compute[i][k], b_frags_compute[j][k]);
                        }
                    }
                }
            }
        }
    }
    __syncthreads();

    for (int i = 0; i < SUB_M_BLOCK / WMMA_M_BLOCK; i++)
    {
        for (int j = 0; j < SUB_N_BLOCK / WMMA_N_BLOCK; j++)
        {
            int block_row = warp_row * (SUB_M_BLOCK / WMMA_M_BLOCK) + i;
            int block_col = warp_col * (SUB_N_BLOCK / WMMA_N_BLOCK) + j;

            int real_row_upper = block_row * WMMA_M_BLOCK + (lane_id >> 2);
            int real_row_lower = real_row_upper + 8;
            int real_col = block_col * WMMA_N_BLOCK + (lane_id % 4) * 2;

            int global_offset_upper = real_row_upper * N_BLOCK + real_col;
            int global_offset_lower = real_row_lower * N_BLOCK + real_col;

            *(uint32_t *)(C + global_offset_upper) = c_frags[i][j][0];
            *(uint32_t *)(C + global_offset_lower) = c_frags[i][j][1];
        }
    }
}

int main(int argc, char *argv[])
{
    int device_id = 0;
    hipDeviceProp_t prop;
    hipSetDevice(device_id);
    hipGetDeviceProperties(&prop, device_id);
    std::cout << "GPU " << prop.pciDeviceID << " Model: " << prop.name << std::endl;

    half *h_A = (half *)malloc(M_BLOCK * K_BLOCK * sizeof(half));
    half *h_B = (half *)malloc(K_BLOCK * N_BLOCK * sizeof(half));

    for (int i = 0; i < M_BLOCK * K_BLOCK; i++)
    {
        h_A[i] = 1;
        h_A[i] = rand() % 2 + 1.0;
    }

    for (int i = 0; i < K_BLOCK * N_BLOCK; i++)
    {
        h_B[i] = 1;
        h_B[i] = rand() % 2 + 1.0;
    }

    half *d_A, *d_B;
    half *d_C;
    hipMalloc(&d_A, M_BLOCK * K_BLOCK * sizeof(half));
    hipMalloc(&d_B, K_BLOCK * N_BLOCK * sizeof(half));
    hipMalloc(&d_C, M_BLOCK * N_BLOCK * sizeof(half));

    hipMemcpy(d_A, h_A, M_BLOCK * K_BLOCK * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K_BLOCK * N_BLOCK * sizeof(half), hipMemcpyHostToDevice);
    hipMemset(d_C, 0, M_BLOCK * N_BLOCK * sizeof(half));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipFuncSetAttribute(reinterpret_cast<const void*>(block_gemm_2d_half_mma), hipFuncAttributeMaxDynamicSharedMemorySize, shmem_size_block_gemm);

    std::cout << "Launching kernel with " << BLOCKS_PER_GRID << " blocks and "
              << NUM_ALLOC_RANK_BLOCK * WARP_SIZE << " threads per block and " << shmem_size_block_gemm << " bytes of shared memory" << std::endl;

    hipEventRecord(start);
    block_gemm_2d_half_mma<<<BLOCKS_PER_GRID, NUM_ALLOC_RANK_BLOCK * WARP_SIZE, shmem_size_block_gemm>>>(d_A, d_B, d_C, 1.0, 0.0);
    hipEventRecord(stop);

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    long long numOpsPerMatrix = 2LL * M_BLOCK * N_BLOCK * K_BLOCK;
    long long totalOps = numOpsPerMatrix * BLOCKS_PER_GRID * NUM_ITER;
    double gflops = static_cast<double>(totalOps) / (milliseconds * 1e6);
    double tflops = gflops / 1000.0f;

    std::cout << "Execution time: " << milliseconds << " ms" << std::endl;
    std::cout << "Performance: " << gflops << " GFLOPS (" << tflops << " TFLOPS)" << std::endl;

    std::cout << "[hemeng_log],2d," << M_BLOCK << "," << NUM_RANK_ALL_BLOCK << "," << NUM_ALLOC_RANK_BLOCK << "," << milliseconds << "," << tflops << std::endl;

    half *h_C = (half *)malloc(M_BLOCK * N_BLOCK * sizeof(half));
    hipMemcpy(h_C, d_C, M_BLOCK * N_BLOCK * sizeof(half), hipMemcpyDeviceToHost);

    half *h_C_ref = (half *)malloc(M_BLOCK * N_BLOCK * sizeof(half));

    for (int i = 0; i < M_BLOCK; i++)
    {
        for (int j = 0; j < M_BLOCK; j++)
        {
            h_C_ref[i * N_BLOCK + j] = 0;
            for (int k = 0; k < K_BLOCK; k++)
            {
                h_C_ref[i * N_BLOCK + j] = (double)h_C_ref[i * N_BLOCK + j] + (double)h_A[i * K_BLOCK + k] * (double)h_B[j * K_BLOCK + k];
            }
        }
    }

    int error = 0;
    for (int i = 0; i < M_BLOCK * N_BLOCK; i++)
    {
        if (fabs((double)h_C[i] / NUM_ITER - (double)h_C_ref[i]) > 1e-6)
        {
            std::cout << "Error: " << i << " " << (double)h_C[i] << " " << (double)h_C_ref[i] << std::endl;
            error = 1;
            break;
        }
    }

    if (!error)
    {
        std::cout << "Validation successful!" << std::endl;
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
