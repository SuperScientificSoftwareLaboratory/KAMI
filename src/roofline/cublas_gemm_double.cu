#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <string.h>

#include <math.h>
#include <iostream>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#define cublasErrCheck(stat)                         \
    {                                                \
        cublasErrCheck_((stat), __FILE__, __LINE__); \
    }

void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line)
{
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
    }
}

__global__ void print_mat_dev(float *A, int m, int n)
{
    for (int i = 0; i < m; i++)
    {
        printf("Row %3d:", i);
        for (int j = 0; j < n; j++)
        {
            printf("%6.2f ", A[i * n + j]);
        }
        printf("\n");
    }
}

int main(int argc, char *argv[])
{

    if (argc < 4)
    {
        printf("Usage: %s <M> <N> <K> [check]\n", argv[0]);
        return 1;
    }

    int M = atoi(argv[1]);
    int N = atoi(argv[2]);
    int K = atoi(argv[3]);

    printf("M = %d, N = %d, K = %d\n", M, N, K);

    double *A = (double *)malloc(M * K * sizeof(double));
    double *B = (double *)malloc(K * N * sizeof(double));

#pragma omp parallel for
    for (int i = 0; i < M * K; i++)
    {
        A[i] = (int)rand() % 100;
    }
#pragma omp parallel for
    for (int i = 0; i < K * N; i++)
    {
        B[i] = (int)rand() % 100;
    }

    double *d_C;
    hipMalloc(&d_C, M * N * sizeof(double));
    hipMemset(d_C, 0, M * N * sizeof(double));

    double *d_A;
    hipMalloc(&d_A, M * K * sizeof(double));
    hipMemcpy(d_A, A, M * K * sizeof(double), hipMemcpyHostToDevice);

    double *d_B;
    hipMalloc(&d_B, K * N * sizeof(double));
    hipMemcpy(d_B, B, K * N * sizeof(double), hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    double alpha = 1.0;
    double beta = 0.0;

    struct timeval start, end;

    hipEvent_t d_start, d_stop;

    hipEventCreate(&d_start);
    hipEventCreate(&d_stop);

    hipDeviceSynchronize();
    gettimeofday(&start, NULL);

    float gemm_time;
    hipEventRecord(d_start, 0);
    cublasErrCheck(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                               N, M, K, &alpha,
                               d_B, N,
                               d_A, K,
                               &beta,
                               d_C, N));
    hipEventRecord(d_stop, 0);
    hipEventSynchronize(d_stop);
    hipEventElapsedTime(&gemm_time, d_start, d_stop);

    hipDeviceSynchronize();
    gettimeofday(&end, NULL);

    hipError_t err = hipGetLastError();

    if (err == hipSuccess)
    {
        double gflops = 2.0 * M * N * K * 1e-9 / ((end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) * 1e-6);
        printf("Size: %d,%d,%d, Time: %f ms, GFLOPS: %f, gemm time: %f\n", M, N, K,
               (end.tv_sec - start.tv_sec) * 1e3 + (end.tv_usec - start.tv_usec) * 1e-3,
               2.0 * M * N * K * 1e-9 / ((end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) * 1e-6),
               gemm_time);
        printf("[hemeng_log]%d,%d,%d,%f,%f,%f\n", M, N, K, (end.tv_sec - start.tv_sec) * 1e3 + (end.tv_usec - start.tv_usec) * 1e-3, 2.0 * M * N * K * 1e-9 / ((end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) * 1e-6), gemm_time);
    }

    return 0;
}