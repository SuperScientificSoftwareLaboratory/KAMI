#include <mma.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define WARP_SIZE 32

using namespace nvcuda;

#ifndef M_BLOCK
#define M_BLOCK 64
#define N_BLOCK 64
#define K_BLOCK 64
#endif

#ifndef NUM_RANK_BLOCK
#define NUM_RANK_BLOCK 2
#endif

#define NUM_RANK_ALL_BLOCK (NUM_RANK_BLOCK * NUM_RANK_BLOCK * NUM_RANK_BLOCK)

#ifndef NUM_ALLOC_RANK_BLOCK
#define NUM_ALLOC_RANK_BLOCK NUM_RANK_ALL_BLOCK
#endif

#define WMMA_M_BLOCK 16
#define WMMA_N_BLOCK 8
#define WMMA_K_BLOCK 8

#define SUB_M_BLOCK (M_BLOCK / NUM_RANK_BLOCK)
#define SUB_N_BLOCK (N_BLOCK / NUM_RANK_BLOCK)
#define SUB_K_BLOCK (K_BLOCK / NUM_RANK_BLOCK / NUM_RANK_BLOCK)

#define NUM_PIPE_M_BLOCK 1
#define NUM_PIPE_N_BLOCK 1
#define NUM_PIPE_K_BLOCK 1

const int shmem_size_block_gemm = (((SUB_M_BLOCK / NUM_PIPE_M_BLOCK) * (SUB_K_BLOCK / NUM_PIPE_K_BLOCK) * NUM_RANK_BLOCK * NUM_RANK_BLOCK) + ((SUB_N_BLOCK / NUM_PIPE_N_BLOCK) * (SUB_K_BLOCK / NUM_PIPE_K_BLOCK) * NUM_RANK_BLOCK * NUM_RANK_BLOCK)) * sizeof(double);

#ifndef NUM_ITER
#define NUM_ITER 1
#endif

#define THREADS_PER_BLOCK (NUM_RANK_ALL_BLOCK * WARP_SIZE)

#ifndef NUM_BATCHES
#define NUM_BATCHES 16384
#endif

__device__ __forceinline__ void mma_m16n8k8(double *acc, const double *frag_a, const double *frag_b)
{
    asm volatile(
        "mma.sync.aligned.m16n8k8.row.col.f64.f64.f64.f64"
        "{%0,  %1,  %2,  %3},"
        "{%4,  %5,  %6,  %7},"
        "{%8,  %9},"
        "{%0,  %1,  %2,  %3};\n"
        : "+d"(acc[0]), "+d"(acc[1]), "+d"(acc[2]), "+d"(acc[3])
        : "d"(frag_a[0]), "d"(frag_a[1]), "d"(frag_a[2]), "d"(frag_a[3]),
          "d"(frag_b[0]), "d"(frag_b[1]));
}

#define FETCH_INT4(pointer) (reinterpret_cast<int4 *>(&(pointer))[0])

__global__ void batched_gemm_3d_double_mma(const double *A, const double *B, double *C, const double alpha, const double beta)
{
    extern __shared__ __align__(16) double shmem[];

    double *shmem_a = shmem;
    double *shmem_b = shmem + (SUB_M_BLOCK / NUM_PIPE_M_BLOCK) * (SUB_K_BLOCK / NUM_PIPE_K_BLOCK) * NUM_RANK_BLOCK * NUM_RANK_BLOCK;

    const unsigned char warp_id = threadIdx.x >> 5;
    const unsigned char lane_id = threadIdx.x & 31;

    const unsigned char warp_id_row = warp_id / NUM_RANK_BLOCK / NUM_RANK_BLOCK;
    const unsigned char warp_id_col = warp_id / NUM_RANK_BLOCK % NUM_RANK_BLOCK;
    const unsigned char warp_id_dep = warp_id % NUM_RANK_BLOCK;

    if (warp_id_row >= NUM_RANK_BLOCK || warp_id_col >= NUM_RANK_BLOCK)
    {
        return;
    }

    const unsigned int block_id = blockIdx.x;
    const double *A_block = A + block_id * (M_BLOCK * K_BLOCK);
    const double *B_block = B + block_id * (K_BLOCK * N_BLOCK);
    double *C_block = C + block_id * (M_BLOCK * N_BLOCK * NUM_RANK_BLOCK);

    const unsigned int dep_offset_a = warp_id_dep * WMMA_M_BLOCK * WMMA_K_BLOCK;
    const unsigned int row_offset_a = warp_id_row * (SUB_M_BLOCK / NUM_PIPE_M_BLOCK) * (SUB_K_BLOCK / NUM_PIPE_K_BLOCK) * NUM_RANK_BLOCK;

    const unsigned int dep_offset_b = warp_id_dep * WMMA_N_BLOCK * WMMA_K_BLOCK;
    const unsigned int col_offset_b = warp_id_col * (SUB_N_BLOCK / NUM_PIPE_N_BLOCK) * (SUB_K_BLOCK / NUM_PIPE_K_BLOCK) * NUM_RANK_BLOCK;

    double a_frags[SUB_M_BLOCK / WMMA_M_BLOCK][SUB_K_BLOCK / WMMA_K_BLOCK][4];
    double b_frags[SUB_N_BLOCK / WMMA_N_BLOCK][SUB_K_BLOCK / WMMA_K_BLOCK][2];
    double a_frags_compute[SUB_M_BLOCK / NUM_PIPE_M_BLOCK / WMMA_M_BLOCK][SUB_K_BLOCK / NUM_PIPE_K_BLOCK / WMMA_K_BLOCK][4];
    double b_frags_compute[SUB_N_BLOCK / NUM_PIPE_N_BLOCK / WMMA_N_BLOCK][SUB_K_BLOCK / NUM_PIPE_K_BLOCK / WMMA_K_BLOCK][2];
    double c_frags[SUB_M_BLOCK / WMMA_M_BLOCK][SUB_N_BLOCK / WMMA_N_BLOCK][4];

    for (int warp_row = 0; warp_row < SUB_M_BLOCK / WMMA_M_BLOCK; warp_row++)
    {
        for (int warp_col = 0; warp_col < SUB_K_BLOCK / WMMA_K_BLOCK; warp_col++)
        {
            int warp_row_offset = warp_id_row * SUB_M_BLOCK + warp_row * WMMA_M_BLOCK;
            int warp_col_offset = (warp_id_col * NUM_RANK_BLOCK + warp_id_dep) * SUB_K_BLOCK + warp_col * WMMA_K_BLOCK;

            int lane_row_upper_offset = lane_id >> 2;
            int lane_row_lower_offset = lane_row_upper_offset + 8;
            int lane_col_left_offset = lane_id % 4;
            int lane_col_right_offset = lane_col_left_offset + 4;

            int global_offset_upper_left = (warp_row_offset + lane_row_upper_offset) * K_BLOCK + warp_col_offset + lane_col_left_offset;
            int global_offset_upper_right = (warp_row_offset + lane_row_upper_offset) * K_BLOCK + warp_col_offset + lane_col_right_offset;
            int global_offset_lower_left = (warp_row_offset + lane_row_lower_offset) * K_BLOCK + warp_col_offset + lane_col_left_offset;
            int global_offset_lower_right = (warp_row_offset + lane_row_lower_offset) * K_BLOCK + warp_col_offset + lane_col_right_offset;

            a_frags[warp_row][warp_col][0] = A_block[global_offset_upper_left];
            a_frags[warp_row][warp_col][1] = A_block[global_offset_lower_left];
            a_frags[warp_row][warp_col][2] = A_block[global_offset_upper_right];
            a_frags[warp_row][warp_col][3] = A_block[global_offset_lower_right];
        }
    }

    for (int warp_row = 0; warp_row < SUB_N_BLOCK / WMMA_N_BLOCK; warp_row++)
    {
        for (int warp_col = 0; warp_col < SUB_K_BLOCK / WMMA_K_BLOCK; warp_col++)
        {
            int warp_row_offset = warp_id_col * SUB_N_BLOCK + warp_row * WMMA_N_BLOCK;
            int warp_col_offset = (warp_id_row * NUM_RANK_BLOCK + warp_id_dep) * SUB_K_BLOCK + warp_col * WMMA_K_BLOCK;

            int lane_row_offset = lane_id >> 2;
            int lane_col_left_offset = lane_id % 4;
            int lane_col_right_offset = lane_col_left_offset + 4;

            int global_offset_left = (warp_row_offset + lane_row_offset) * K_BLOCK + warp_col_offset + lane_col_left_offset;
            int global_offset_right = (warp_row_offset + lane_row_offset) * K_BLOCK + warp_col_offset + lane_col_right_offset;

            b_frags[warp_row][warp_col][0] = B_block[global_offset_left];
            b_frags[warp_row][warp_col][1] = B_block[global_offset_right];
        }
    }

    for (int i = 0; i < SUB_M_BLOCK / WMMA_M_BLOCK; ++i)
    {
        for (int j = 0; j < SUB_N_BLOCK / WMMA_N_BLOCK; ++j)
        {
            c_frags[i][j][0] = 0.0;
            c_frags[i][j][1] = 0.0;
            c_frags[i][j][2] = 0.0;
            c_frags[i][j][3] = 0.0;
        }
    }

    __syncthreads();

    for (int idx_iter = 0; idx_iter < NUM_ITER; idx_iter++)
    {
        for (int idx_stage = 0; idx_stage < NUM_RANK_BLOCK; idx_stage++)
        {
            for (int idx_pipe_k = 0; idx_pipe_k < NUM_PIPE_K_BLOCK; idx_pipe_k++)
            {
#pragma unroll
                for (int idx_pipe_n = 0; idx_pipe_n < NUM_PIPE_N_BLOCK; idx_pipe_n++)
                {
                    for (int idx_pipe_m = 0; idx_pipe_m < NUM_PIPE_M_BLOCK; idx_pipe_m++)
                    {
                        if (warp_id_col == idx_stage)
                        {
#pragma unroll
                            for (int i = 0; i < (SUB_M_BLOCK / NUM_PIPE_M_BLOCK) / WMMA_M_BLOCK; ++i)
                            {
#pragma unroll
                                for (int j = 0; j < SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_K_BLOCK; ++j)
                                {
                                    int real_i = i + idx_pipe_m * (SUB_M_BLOCK / WMMA_M_BLOCK / NUM_PIPE_M_BLOCK);
                                    int real_j = j + idx_pipe_k * (SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_K_BLOCK);

                                    int smem_offset = lane_id * 4 + (i * (SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_K_BLOCK) + j) * NUM_RANK_BLOCK * WMMA_M_BLOCK * WMMA_K_BLOCK + dep_offset_a + row_offset_a;

                                    shmem_a[smem_offset] = a_frags[real_i][real_j][0];
                                    shmem_a[smem_offset + 1] = a_frags[real_i][real_j][1];
                                    shmem_a[smem_offset + 2] = a_frags[real_i][real_j][2];
                                    shmem_a[smem_offset + 3] = a_frags[real_i][real_j][3];
                                }
                            }
                        }

                        if (warp_id_row == idx_stage)
                        {
#pragma unroll
                            for (int i = 0; i < (SUB_N_BLOCK / NUM_PIPE_N_BLOCK) / WMMA_N_BLOCK; ++i)
                            {
#pragma unroll
                                for (int j = 0; j < SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_K_BLOCK; ++j)
                                {
                                    int real_i = i + idx_pipe_n * (SUB_N_BLOCK / WMMA_N_BLOCK / NUM_PIPE_N_BLOCK);
                                    int real_j = j + idx_pipe_k * (SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_K_BLOCK);

                                    int smem_offset = lane_id * 2 + (i * (SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_K_BLOCK) + j) * (NUM_RANK_BLOCK * WMMA_N_BLOCK * WMMA_K_BLOCK) + dep_offset_b + col_offset_b;

                                    shmem_b[smem_offset] = b_frags[real_i][real_j][0];
                                    shmem_b[smem_offset + 1] = b_frags[real_i][real_j][1];
                                }
                            }
                        }

                        __syncthreads();

#pragma unroll
                        for (int i = 0; i < (SUB_M_BLOCK / NUM_PIPE_M_BLOCK) / WMMA_M_BLOCK; ++i)
                        {
#pragma unroll
                            for (int j = 0; j < SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_K_BLOCK; ++j)
                            {

                                int smem_offset = lane_id * 4 + (i * (SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_K_BLOCK) + j) * NUM_RANK_BLOCK * WMMA_M_BLOCK * WMMA_K_BLOCK + dep_offset_a + row_offset_a;

                                a_frags_compute[i][j][0] = shmem_a[smem_offset];
                                a_frags_compute[i][j][1] = shmem_a[smem_offset + 1];
                                a_frags_compute[i][j][2] = shmem_a[smem_offset + 2];
                                a_frags_compute[i][j][3] = shmem_a[smem_offset + 3];
                            }
                        }

#pragma unroll
                        for (int i = 0; i < (SUB_N_BLOCK / NUM_PIPE_N_BLOCK) / WMMA_N_BLOCK; ++i)
                        {
#pragma unroll
                            for (int j = 0; j < SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_K_BLOCK; ++j)
                            {

                                int smem_offset = lane_id * 2 + (i * (SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_K_BLOCK) + j) * (NUM_RANK_BLOCK * WMMA_N_BLOCK * WMMA_K_BLOCK) + dep_offset_b + col_offset_b;

                                b_frags_compute[i][j][0] = shmem_b[smem_offset];
                                b_frags_compute[i][j][1] = shmem_b[smem_offset + 1];
                            }
                        }

                        __syncthreads();

#pragma unroll
                        for (int i = 0; i < (SUB_M_BLOCK / NUM_PIPE_M_BLOCK) / WMMA_M_BLOCK; i++)
                        {
#pragma unroll
                            for (int j = 0; j < (SUB_N_BLOCK / NUM_PIPE_N_BLOCK) / WMMA_N_BLOCK; j++)
                            {
#pragma unroll
                                for (int k = 0; k < SUB_K_BLOCK / WMMA_K_BLOCK / NUM_PIPE_K_BLOCK; k++)
                                {

                                    int real_i = i + idx_pipe_m * (SUB_M_BLOCK / WMMA_M_BLOCK / NUM_PIPE_M_BLOCK);
                                    int real_j = j + idx_pipe_n * (SUB_N_BLOCK / WMMA_N_BLOCK / NUM_PIPE_N_BLOCK);
                                    mma_m16n8k8(c_frags[real_i][real_j], a_frags_compute[i][k], b_frags_compute[j][k]);
                                }
                            }
                        }
                    }
                }
            }
        }
    }

    __syncthreads();

    for (int warp_row = 0; warp_row < SUB_M_BLOCK / WMMA_M_BLOCK; warp_row++)
    {
        for (int warp_col = 0; warp_col < SUB_N_BLOCK / WMMA_N_BLOCK; warp_col++)
        {
            int warp_row_offset = warp_id_row * SUB_M_BLOCK + warp_row * WMMA_M_BLOCK;
            int warp_col_offset = warp_id_col * SUB_N_BLOCK + warp_col * WMMA_N_BLOCK;

            int lane_row_upper_offset = lane_id >> 2;
            int lane_row_lower_offset = lane_row_upper_offset + 8;
            int lane_col_offset = (lane_id % 4) * 2;

            int global_offset_upper = (warp_row_offset + lane_row_upper_offset) * N_BLOCK + warp_col_offset + lane_col_offset + warp_id_dep * M_BLOCK * N_BLOCK;
            int global_offset_lower = (warp_row_offset + lane_row_lower_offset) * N_BLOCK + warp_col_offset + lane_col_offset + warp_id_dep * M_BLOCK * N_BLOCK;

            C_block[global_offset_upper] = c_frags[warp_row][warp_col][0];
            C_block[global_offset_upper + 1] = c_frags[warp_row][warp_col][1];
            C_block[global_offset_lower] = c_frags[warp_row][warp_col][2];
            C_block[global_offset_lower + 1] = c_frags[warp_row][warp_col][3];
        }
    }
}

int main(int argc, char *argv[])
{
    int device_id = 0;
    hipDeviceProp_t prop;
    hipSetDevice(device_id);
    hipGetDeviceProperties(&prop, device_id);
    std::cout << "GPU " << prop.pciDeviceID << " Model: " << prop.name << std::endl;

    double *h_A = (double *)malloc(sizeof(double) * M_BLOCK * K_BLOCK * NUM_BATCHES);
    double *h_B = (double *)malloc(sizeof(double) * K_BLOCK * N_BLOCK * NUM_BATCHES);

#pragma omp parallel for
    for (unsigned long i = 0; i < (unsigned long)M_BLOCK * K_BLOCK * NUM_BATCHES; i++)
    {
        h_A[i] = 1;
        unsigned int local_seed = i;
        h_A[i] = rand_r(&local_seed) % 3;
    }

#pragma omp parallel for
    for (unsigned long i = 0; i < (unsigned long)K_BLOCK * N_BLOCK * NUM_BATCHES; i++)
    {
        h_B[i] = 1;
        unsigned int local_seed = i;
        h_B[i] = rand_r(&local_seed) % 3;
    }

    double *d_A, *d_B;
    double *d_C;
    hipMalloc(&d_A, sizeof(double) * M_BLOCK * K_BLOCK * NUM_BATCHES);
    hipMalloc(&d_B, sizeof(double) * K_BLOCK * N_BLOCK * NUM_BATCHES);
    hipMalloc(&d_C, sizeof(double) * M_BLOCK * N_BLOCK * NUM_BATCHES * NUM_RANK_BLOCK);

    hipMemcpy(d_A, h_A, sizeof(double) * M_BLOCK * K_BLOCK * NUM_BATCHES, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(double) * K_BLOCK * N_BLOCK * NUM_BATCHES, hipMemcpyHostToDevice);
    hipMemset(d_C, 0, sizeof(double) * M_BLOCK * N_BLOCK * NUM_BATCHES * NUM_RANK_BLOCK);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipFuncSetAttribute(reinterpret_cast<const void*>(batched_gemm_3d_double_mma), hipFuncAttributeMaxDynamicSharedMemorySize, shmem_size_block_gemm);

    std::cout << "Launching kernel with " << NUM_BATCHES << " blocks and "
              << NUM_ALLOC_RANK_BLOCK * WARP_SIZE << " threads per block and " << shmem_size_block_gemm << " bytes of shared memory" << std::endl;

    hipEventRecord(start);
    batched_gemm_3d_double_mma<<<NUM_BATCHES, NUM_ALLOC_RANK_BLOCK * WARP_SIZE, shmem_size_block_gemm>>>(d_A, d_B, d_C, 1.0, 0.0);
    hipDeviceSynchronize();
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    long long numOpsPerMatrix = 2LL * M_BLOCK * N_BLOCK * K_BLOCK;
    long long totalOps = numOpsPerMatrix * NUM_BATCHES * NUM_ITER;
    double gflops = static_cast<double>(totalOps) / (milliseconds * 1e6);
    double tflops = gflops / 1000.0f;

    std::cout << "Execution time: " << milliseconds << " ms" << std::endl;
    std::cout << "Performance: " << gflops << " GFLOPS (" << tflops << " TFLOPS)" << std::endl;
    std::cout << "[hemeng_log],3d," << M_BLOCK << "," << N_BLOCK << "," << K_BLOCK << "," << NUM_BATCHES << "," << tflops << "," << THREADS_PER_BLOCK << std::endl;

    double *h_C = (double *)malloc(sizeof(double) * M_BLOCK * N_BLOCK * NUM_BATCHES * NUM_RANK_BLOCK);
    hipMemcpy(h_C, d_C, sizeof(double) * M_BLOCK * N_BLOCK * NUM_BATCHES * NUM_RANK_BLOCK, hipMemcpyDeviceToHost);

#pragma omp parallel for
    for (unsigned long batch = 0; batch < NUM_BATCHES; batch++)
    {
        for (int i = 0; i < M_BLOCK; i++)
        {
            for (int j = 0; j < N_BLOCK; j++)
            {
                for (int k = 1; k < NUM_RANK_BLOCK; k++)
                {
                    h_C[i * N_BLOCK + j + batch * M_BLOCK * N_BLOCK * NUM_RANK_BLOCK] += h_C[i * N_BLOCK + j + k * M_BLOCK * N_BLOCK + batch * M_BLOCK * N_BLOCK * NUM_RANK_BLOCK];
                }
            }
        }
    }

    double *h_C_ref = (double *)malloc(sizeof(double) * M_BLOCK * N_BLOCK * NUM_BATCHES);

#pragma omp parallel for
    for (unsigned long batch = 0; batch < NUM_BATCHES; batch++)
    {
        for (unsigned long i = 0; i < M_BLOCK; i++)
        {
            for (unsigned long j = 0; j < M_BLOCK; j++)
            {
                h_C_ref[i * N_BLOCK + j + batch * M_BLOCK * N_BLOCK] = 0;
                for (unsigned long k = 0; k < K_BLOCK; k++)
                {
                    h_C_ref[i * N_BLOCK + j + batch * M_BLOCK * N_BLOCK] += h_A[i * K_BLOCK + k + batch * M_BLOCK * K_BLOCK] * h_B[j * K_BLOCK + k + batch * N_BLOCK * K_BLOCK];
                }
            }
        }
    }

    int error = 0;
#pragma omp parallel for
    for (unsigned long i = 0; i < (unsigned long)NUM_BATCHES; i++)
    {
        for (int j = 0; j < M_BLOCK * N_BLOCK; j++)
        {
            if (fabs(h_C[j + i * M_BLOCK * N_BLOCK * NUM_RANK_BLOCK] / NUM_ITER - h_C_ref[j + i * M_BLOCK * N_BLOCK]) > 1e-6)
            {

                error = 1;
            }
        }
    }

    if (!error)
    {
        std::cout << "Validation successful!" << std::endl;
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
